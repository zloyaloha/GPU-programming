
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#define CSC(call)  									                \
do {											                    \
	hipError_t res = call;							                \
	if (res != hipSuccess) {							            \
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);								                    \
	}										                        \
} while(0)

__global__ void kernel(hipTextureObject_t tex, uchar4 *out, int w, int h) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
   	int offset_x = blockDim.x * gridDim.x;
	int offset_y = blockDim.y * gridDim.y;
    int x, y;
    uchar4 p;
    for(y = idy; y < h; y += offset_y)
		for(x = idx; x < w; x += offset_x) {
            p = tex2D< uchar4 >(tex, double(x) / w, double(y) / h);
            out[y * w + x] = make_uchar4(255 - p.x, 255 - p.y, 255 - p.z, p.w);
        }
}

int main() {
    int w, h;
   	FILE *fp = fopen("in.bin", "rb");
    if (!fp) {
        std::cerr << "No such file" << std::endl;
        return 0;
    }
 	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
    uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    fread(data, sizeof(uchar4), w * h, fp);
    fclose(fp);

    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpy2DToArray(arr, 0, 0, data, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeMirror; // Clamp
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = true;

    hipTextureObject_t tex = 0;
    CSC(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

    uchar4 *dev_out;
	CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

    kernel<<< dim3(16, 16), dim3(32, 32) >>>(tex, dev_out, w, h);
    CSC(hipGetLastError());

    CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

	CSC(hipDestroyTextureObject(tex));
	CSC(hipFreeArray(arr));
	CSC(hipFree(dev_out));

    fp = fopen("out.bin", "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

    free(data);
    return 0;
}